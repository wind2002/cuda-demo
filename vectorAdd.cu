#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int N = 1000; // Vector size
    size_t size = N * sizeof(float);

    // Host memory
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy inputs to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < N; i++) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            printf("Verification failed at index %d!\n", i);
            break;
        }
    }
    printf("Vector addition completed successfully!\n");

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}