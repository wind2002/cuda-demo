#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    // Step 1: Allocate memory for arrays on the CPU and GPU
    const int N = 1000; // Vector size
    size_t size = N * sizeof(float);

    // Allocate memory for arrays on the CPU (host)
    float *h_A = (float *)malloc(size); // Input vector A
    float *h_B = (float *)malloc(size); // Input vector B
    float *h_C = (float *)malloc(size); // Output vector C

    // Step 2: Initialize two input arrays with random numbers
    for (int i = 0; i < N; i++) {
        h_A[i] = rand() / (float)RAND_MAX; // Random values between 0 and 1
        h_B[i] = rand() / (float)RAND_MAX; // Random values between 0 and 1
    }

    // Allocate memory for arrays on the GPU (device)
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size); // GPU memory for input vector A
    hipMalloc(&d_B, size); // GPU memory for input vector B
    hipMalloc(&d_C, size); // GPU memory for output vector C

    // Step 3: Copy arrays to the GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); // Copy A from CPU to GPU
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice); // Copy B from CPU to GPU

    // Step 4: Launch the vectorAdd kernel to perform addition
    int threadsPerBlock = 256; // Number of threads per block
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; // Number of blocks
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N); // Launch kernel

    // Step 5: Copy the result back to the CPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost); // Copy result C from GPU to CPU

    // Step 6: Verify the result by checking if C[i] = A[i] + B[i] for all elements
    for (int i = 0; i < N; i++) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            printf("Verification failed at index %d!\n", i);
            break;
        }
    }
    printf("Vector addition completed successfully!\n");

    // Step 7: Free memory
    hipFree(d_A); // Free GPU memory for A
    hipFree(d_B); // Free GPU memory for B
    hipFree(d_C); // Free GPU memory for C
    free(h_A);     // Free CPU memory for A
    free(h_B);     // Free CPU memory for B
    free(h_C);     // Free CPU memory for C
